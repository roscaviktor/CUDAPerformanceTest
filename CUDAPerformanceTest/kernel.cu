#include "hip/hip_runtime.h"
/***********************************************************************
 Author: Victor Rosca
 Date: 2016-02-10

                     CUDA. Testing Performance.
 
 This application generate an image using CUDA. 
 CUDA is a parallel computing platform and application programming interface (API) model created by NVIDIA.
 It allows software developers to use a CUDA-enabled graphics processing unit (GPU) for general purpose 
 processing � an approach known as GPGPU. The CUDA platform is a software layer that gives direct access 
 to the GPU's virtual instruction set and parallel computational elements.

 ************************************************************************/


#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

////////////////////////////////////////////////////////////////////
//#define IMAGE_TYPE_1
//#define IMAGE_TYPE_2
#define IMAGE_TYPE_3
//#define IMAGE_TYPE_4

#define IMAGE_WIDTH		5000
#define IMAGE_HEIGHT	5000
#define IMAGE_LEN		IMAGE_WIDTH * IMAGE_HEIGHT

// If is defined TESTING_PERFORMANCE then the program will generate a HTML file that will contain the result of testing. 
#define TESTING_PERFORMANCE

#ifdef TESTING_PERFORMANCE
#define BLOCK_STEP 10
#define INITIAL_BLOCKS 400
#define FINAL_BLOCKS IMAGE_LEN
#define MAX_IMAGES_GENERATED 2
#endif

hipError_t generateImage(float *time, int calledNo);

#ifndef TESTING_PERFORMANCE
////////////////////////////////////////////////////////////////////
bool drawBMP(char *filename, char *blue, char *green, char *red) {
	unsigned int headers[13];
	FILE * outfile;
	int extrabytes;
	int paddedsize;
	int x; int y; int n;

	// How many bytes of padding to add to each
	extrabytes = 4 - ((IMAGE_WIDTH * 3) % 4);                
	// horizontal line - the size of which must
	// be a multiple of 4 bytes.
	if (extrabytes == 4)
		extrabytes = 0;

	paddedsize = ((IMAGE_WIDTH * 3) + extrabytes) * IMAGE_HEIGHT;

	// Headers...
	// Note that the "BM" identifier in bytes 0 and 1 is NOT included in these "headers".
	headers[0] = paddedsize + 54;			// bfSize (whole file size)
	headers[1] = 0;							// bfReserved (both)
	headers[2] = 54;						// bfOffbits
	headers[3] = 40;						// biSize
	headers[4] = IMAGE_WIDTH;				// biWidth
	headers[5] = IMAGE_HEIGHT;				// biHeight

	// Would have biPlanes and biBitCount in position 6, but they're shorts.
	// It's easier to write them out separately (see below) than pretend
	// they're a single int, especially with endian issues...
	headers[7] = 0;							// biCompression
	headers[8] = paddedsize;				// biSizeImage
	headers[9] = 0;							// biXPelsPerMeter
	headers[10] = 0;						// biYPelsPerMeter
	headers[11] = 0;						// biClrUsed
	headers[12] = 0;						// biClrImportant

	outfile = fopen(filename, "wb");

	// Headers begin...
	// When printing ints and shorts, we write out 1 character at a time to avoid endian issues.
	fprintf(outfile, "BM");
	for (n = 0; n <= 5; n++){
		fprintf(outfile, "%c", headers[n] & 0x000000FF);
		fprintf(outfile, "%c", (headers[n] & 0x0000FF00) >> 8);
		fprintf(outfile, "%c", (headers[n] & 0x00FF0000) >> 16);
		fprintf(outfile, "%c", (headers[n] & (unsigned int)0xFF000000) >> 24);
	}

	// These next 4 characters are for the biPlanes and biBitCount fields.
	fprintf(outfile, "%c", 1);
	fprintf(outfile, "%c", 0);
	fprintf(outfile, "%c", 24);
	fprintf(outfile, "%c", 0);

	for (n = 7; n <= 12; n++){
		fprintf(outfile, "%c", headers[n] & 0x000000FF);
		fprintf(outfile, "%c", (headers[n] & 0x0000FF00) >> 8);
		fprintf(outfile, "%c", (headers[n] & 0x00FF0000) >> 16);
		fprintf(outfile, "%c", (headers[n] & (unsigned int)0xFF000000) >> 24);
	}

	// Headers done, now write the data...
	int i;

	// BMP image format is written from bottom to top...
	for (y = IMAGE_HEIGHT - 1; y >= 0; y--){		
		for (x = 0; x <= IMAGE_WIDTH - 1; x++){
			i = y * x;			
			// Also, it's written in (b,g,r) format...
			fprintf(outfile, "%c", blue[i]);
			fprintf(outfile, "%c", green[i]);
			fprintf(outfile, "%c", red[i]);
		}

		if (extrabytes){
			// See above - BMP lines must be of lengths divisible by 4.
			for (n = 1; n <= extrabytes; n++){
				fprintf(outfile, "%c", 0);
			}
		}
	}
	fclose(outfile);
	return true;
}
#endif

////////////////////////////////////////////////////////////////////
char *time_stamp(){
	char *timestamp = (char *)malloc(sizeof(char)* 16);
	time_t ltime;
	ltime = time(NULL);
	struct tm *tm;
	tm = localtime(&ltime);

	sprintf(timestamp, "%04d%02d%02d%02d%02d%02d", tm->tm_year + 1900, tm->tm_mon,
		tm->tm_mday, tm->tm_hour, tm->tm_min, tm->tm_sec);
	return timestamp;
}



////////////////////////////////////////////////////////////////////
// This function is running in GPU.
__global__ void runKernel(char *blue, char *green, char *red, long blockSize, long width, long height)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
#ifdef IMAGE_TYPE_1
	blue[i] = (char)((255));
	green[i] = (char)((255));
	red[i] = (char)(((width*height) / i) % 256);
#endif
#ifdef IMAGE_TYPE_2
	blue[i] = (char)(width * height / i * 256) % 256;
	green[i] = (char)(width * height / i * 128) % 256;
	red[i] = (char)(width * height / i * 64) % 256;
#endif
#ifdef IMAGE_TYPE_3
	blue[i] = (char)((i * 256) / (width * height));
	green[i] = 64;
	red[i] = 64;
#endif
#ifdef IMAGE_TYPE_4
	long y = i / width;
	switch (threadIdx.x % 3){
	case 0:
		blue[i] = 255;
		green[i] = 0;
		red[i] = 0;
		//blue[i] = (char)(blockIdx.x * 256 / blockDim.x);
		//green[i] = (char)(blockIdx.x * 64 / blockDim.x);
		//red[i] = (char)(blockIdx.x * 64 / blockDim.x);
		break;
	case 1:
		blue[i] = 0;
		green[i] = 255;
		red[i] = 0;
		//blue[i] = (char)(blockIdx.x * 64 / blockDim.x);
		//green[i] = (char)(blockIdx.x * 256 / blockDim.x);
		//red[i] = (char)(blockIdx.x * 64 / blockDim.x);
		break;
	case 2:
		blue[i] = 0;
		green[i] = 0;
		red[i] = 255;
		//blue[i] = (char)(blockIdx.x * 64 / blockDim.x);
		//green[i] = (char)(blockIdx.x * 64 / blockDim.x);
		//red[i] = (char)(blockIdx.x * 256 / blockDim.x);
		break;
	}
#endif
}

////////////////////////////////////////////////////////////////////
// Helper function for using CUDA to add vectors in parallel.
hipError_t generateImage(float *time, int calledNo)
{
	char *dev_blue = 0;
	char *dev_green = 0;
	char *dev_red = 0;
    hipError_t cudaStatus;

	char *blue = nullptr;
	char *green = nullptr;
	char *red = nullptr;

	blue = new char[IMAGE_LEN];
	green = new char[IMAGE_LEN];
	red = new char[IMAGE_LEN];

	//long start;
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_blue, IMAGE_LEN * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_green, IMAGE_LEN * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_red, IMAGE_LEN * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel.
	long blockSize = IMAGE_LEN / calledNo;

	hipEventRecord(start);
	runKernel << <blockSize, calledNo >> >(dev_blue, dev_green, dev_red, blockSize, IMAGE_WIDTH, IMAGE_HEIGHT);
	hipEventRecord(stop);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "runKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching runKernel!\n", cudaStatus);
        goto Error;
    }

	hipEventSynchronize(stop);
	hipEventElapsedTime(time, start, stop);

    // Deploy from GPU buffer to host memory.
#ifndef TESTING_PERFORMANCE
	cudaStatus = hipMemcpy(blue, dev_blue, IMAGE_LEN * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!"); goto Error;
	}
	cudaStatus = hipMemcpy(green, dev_green, IMAGE_LEN * sizeof(char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!"); goto Error;
	}
	cudaStatus = hipMemcpy(red, dev_red, IMAGE_LEN * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!"); goto Error;
    }

	// Create a BMP file.
	char fileStr[25];
	sprintf(fileStr, "%s.bmp", time_stamp());
	if (drawBMP(fileStr, blue, green, red)){
		printf("Image was printed!\n");
	}
#endif

Error:
	hipFree(dev_blue);
	hipFree(dev_green);
	hipFree(dev_red);
	delete blue;
	delete green;
	delete red;
    
    return cudaStatus;
}


////////////////////////////////////////////////////////////////////
int main()
{
	float time = 0.0;

#ifdef TESTING_PERFORMANCE
	int blocks = INITIAL_BLOCKS;
	FILE *file;
	hipError_t cudaStatus;
	char fileStr[25];
	sprintf(fileStr, "%s.html", time_stamp());
	file = fopen(fileStr, "wb");
	fprintf(file, "<html>\n \
	<head>\n \
		<title>CUDA - Testing Performance, Victor Rosca</title>\n \
		<script type = \"text/javascript\" src = \"https://www.gstatic.com/charts/loader.js\"></script>\n \
		<script type = \"text/javascript\">\n \
		google.charts.load('current', { 'packages':['corechart'] });\n \
	google.charts.setOnLoadCallback(drawChart);\n \
	\n\
	function drawChart() { \n\
		var data = google.visualization.arrayToDataTable([ \n\
			['Time (msec)', 'Generate %d image(s).'], \n\
				", MAX_IMAGES_GENERATED);
	printf("Testing CUDA. \n\n", blocks, time);
	printf("Image size: ~ %lu Mb\n\n", (IMAGE_LEN * 3 / 1024 / 1024));
	printf("-------------------------------------------\n");
	printf("|  Blocks of threads |     Time (msec)    |\n");
	printf("|-----------------------------------------|\n");
	while (blocks < FINAL_BLOCKS){
		long blockSize = IMAGE_LEN / blocks;
		int i = 0;
		bool stop = false;
		float totalTime = 0.0;
		while (i++ < MAX_IMAGES_GENERATED){
			cudaStatus = generateImage(&time, blocks);
			totalTime += time;
			if (cudaStatus != hipSuccess){
				stop = true;
				break;
			}
		}
		if (stop)
			break;
		fprintf(file, "['<<<%d, %d>>>', %f],\n", blocks, blockSize, totalTime);
		printf("|%*d|%*f|\n", 20, blocks, 20, totalTime);
		blocks += BLOCK_STEP;
	}
	printf("-------------------------------------------\n");

	fprintf(file, "]); \n\
		var options = { \n\
		title: 'CUDA Performance, file: %s, author: Victor Rosca', \n\
		   curveType : 'function', \n\
				   legend : { position: 'bottom' } \n\
		}; \n\
		var chart = new google.visualization.LineChart(document.getElementById('curve_chart')); \n\
		chart.draw(data, options); \n\
	} \n\
	</script> \n\
		</head> \n\
		<body> \n\
			<p>Image size: ~ %lu Mb, </br> \n\
			<p>Initial blocks of threads = %d</br> \n\
			Final blocks of threads = %d</p> \n\
		<div id = \"curve_chart\" style = \"width: 1200px; height: 700px\"></div> \n\
		</body> \n\
	</html>", fileStr, (long)(IMAGE_LEN * 3 / 1024 / 1024), (int)INITIAL_BLOCKS, blocks);
	
	fclose(file);

	if (cudaStatus != hipSuccess){
		fprintf(stderr, "generateImage failed!");
		printf("END. Press any key.\n");
		getch();
		return 1;
	}
#else
	hipError_t cudaStatus = generateImage(&time, 1000);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "generateImage failed!");
		printf("END. Press any key.\n");
		getch();
		return 1;
	}
#endif

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		printf("END. Press any key.\n");
		getch();
		return 1;
	}
	printf("END. Press any key.\n");
	getch();
	return 0;
}
